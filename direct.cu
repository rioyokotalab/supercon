#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define THREADS 512

double get_time() {
  struct timeval tv;
  gettimeofday(&tv,NULL);
  return (double)(tv.tv_sec+tv.tv_usec*1e-6);
}

__global__ void GPUkernel(int N, double * x, double * y, double * z, double * m,
			  double * ax, double * ay, double * az, double G, double eps) {
  int i = blockIdx.x * THREADS + threadIdx.x;
  double axi = 0;
  double ayi = 0;
  double azi = 0;
  double xi = x[i];
  double yi = y[i];
  double zi = z[i];
  double Gmi = G * m[i];
  __shared__ double xj[THREADS], yj[THREADS], zj[THREADS], mj[THREADS];
  for ( int jb=0; jb<N/THREADS; jb++ ) {
    __syncthreads();
    xj[threadIdx.x] = x[jb*THREADS+threadIdx.x];
    yj[threadIdx.x] = y[jb*THREADS+threadIdx.x];
    zj[threadIdx.x] = z[jb*THREADS+threadIdx.x];
    mj[threadIdx.x] = m[jb*THREADS+threadIdx.x];
    __syncthreads();
#pragma unroll
    for( int j=0; j<THREADS; j++ ) {
      double dx = xi - xj[j];
      double dy = yi - yj[j];
      double dz = zi - zj[j];
      double R2 = dx * dx + dy * dy + dz * dz + eps;
      double invR = rsqrtf(R2);
      double invR3 = invR * invR * invR * Gmi * mj[j];
      axi -= dx * invR3;
      ayi -= dy * invR3;
      azi -= dz * invR3;
    }
  }
  ax[i] = axi;
  ay[i] = ayi;
  az[i] = azi;
}

int main() {
// Initialize
  int N = 1 << 16;
  int i, j;
  double OPS = 20. * N * N * 1e-9;
  double G = 6.6743e-11;
  double EPS = 1e-4;
  double tic, toc;
  double *x, *y, *z, *m, *ax, *ay, *az;
  hipMallocManaged((void**)&x, N * sizeof(double));
  hipMallocManaged((void**)&y, N * sizeof(double));
  hipMallocManaged((void**)&z, N * sizeof(double));
  hipMallocManaged((void**)&m, N * sizeof(double));
  hipMallocManaged((void**)&ax, N * sizeof(double));
  hipMallocManaged((void**)&ay, N * sizeof(double));
  hipMallocManaged((void**)&az, N * sizeof(double));
  for (i=0; i<N; i++) {
    x[i] = drand48();
    y[i] = drand48();
    z[i] = drand48();
    m[i] = drand48() / N;
  }
  printf("N      : %d\n",N);

// CUDA
  tic = get_time();
  GPUkernel<<<N/THREADS,THREADS>>>(N, x, y, z, m, ax, ay, az, G, EPS);
  hipDeviceSynchronize();
  toc = get_time();
  printf("CUDA   : %e s : %lf GFlops\n",toc-tic, OPS/(toc-tic));

// No CUDA
  double diff = 0, norm = 0;
  tic = get_time();
#pragma omp parallel for private(j) reduction(+: diff, norm)
  for (i=0; i<N; i++) {
    double axi = 0;
    double ayi = 0;
    double azi = 0;
    double Gmi = G * m[i];
    for (j=0; j<N; j++) {
      double dx = x[i] - x[j];
      double dy = y[i] - y[j];
      double dz = z[i] - z[j];
      double R2 = dx * dx + dy * dy + dz * dz + EPS;
      double invR = 1.0f / sqrtf(R2);
      double invR3 = invR * invR * invR * Gmi * m[j];
      axi -= dx * invR3;
      ayi -= dy * invR3;
      azi -= dz * invR3;
    }
    diff += (ax[i] - axi) * (ax[i] - axi)
      + (ay[i] - ayi) * (ay[i] - ayi)
      + (az[i] - azi) * (az[i] - azi);
    norm += axi * axi + ayi * ayi + azi * azi;    
  }
  toc = get_time();
  printf("No CUDA: %e s : %lf GFlops\n",toc-tic, OPS/(toc-tic));
  printf("Error  : %e\n",sqrt(diff/norm));

// DEALLOCATE
  hipFree(x);
  hipFree(y);
  hipFree(z);
  hipFree(m);
  hipFree(ax);
  hipFree(ay);
  hipFree(az);
  return 0;
}
